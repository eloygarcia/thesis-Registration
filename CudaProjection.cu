#include "hip/hip_runtime.h"
#include "CudaProjection.h"
#include "CudaLibrary_kernels.cu"

#include "hip/hip_runtime.h"
#include ""

// Constructor y Destructor

CudaProjection::CudaProjection()
{
	// mri
	m_mri_size = new int[3];
	m_mri_spacing = new float[3];
	m_mri_origen = new float[3];
//	m_mri_imagepointer = 0;

	// mesh and grid !
//	m_i_points = 0;
//	m_f_points = 0;
//	m_elements = 0;

	m_grid_origen = new float[3];
	m_grid_spacing = new float[3];
	m_grid_size = new int[3];
	
//	m_flags = 0;
//	m_cumsum = 0;
//	m_correspondingElements = 0;

	// mamo simulada
	m_simulada_size = new int[3];
	m_simulada_spacing = new float[3];
	m_simulada_origen = new float[3];
	//m_2d_imagepointer = = new float[ numberOfPixels_2D ];

	m_source = new float[3];

	// Variables en:  device
//	dev_mri_size = new int[3];
//	dev_mri_spacing = new float[3];
//	dev_mri_origen = new float[3];
//numberOfPixels_3D = 0 ;
//	dev_3d_imagepointer = new float[ numberOfPixels_3D ];

//	dev_simulada_size = new int[3];
//	dev_simulada_spacing = new float[3];
//	dev_simulada_origen = new float[3]; 
//numberOfPixels_2D = ( m_2d_size[0] * m_2d_size[1] );
//	dev_2d_imagepointer = new float[ numberOfPixels_2D ];

//	dev_source = new float[3];

	hipError_t cudaStatus;
}

CudaProjection::~CudaProjection()
{
	// delete [] m_parameters;
	// host
	delete[] m_mri_size;
	delete[] m_mri_spacing;
	delete[] m_mri_origen;
	delete[] m_mri_imagepointer;

	delete [] m_i_points;
	delete [] m_f_points;
	delete [] m_elements;

	delete [] m_grid_origen;
	delete [] m_grid_size;
	delete [] m_grid_spacing;

	delete [] m_flags;
	delete [] m_cumsum;
	delete [] m_correspondingElements;

	delete[] m_simulada_size;
	delete[] m_simulada_spacing;
	delete[] m_simulada_origen;
//	delete[] m_simulada_imagepointer;

	delete[] m_source;

	// device
//	delete[] dev_mri_size;
//	delete[] dev_mri_spacing;
//	delete[] dev_mri_origen;
//	delete[] dev_mri_imagepointer;
//
//	delete [] dev_i_points;
//	delete [] dev_f_points;
//	delete [] dev_elements;
//
//	delete [] dev_grid_origen;
//	delete [] dev_grid_size;
//	delete [] dev_grid_spacing;
//
//	delete [] dev_flags;
//	delete [] dev_cumsum;
//	delete [] dev_correspondingElements;
//
//	delete[] dev_simulada_size;
//	delete[] dev_simulada_spacing;
//	delete[] dev_simulada_origen;
//	delete[] dev_simulada_imagepointer;
//
//	delete[] dev_source;
}

void CudaProjection::Initialize()
{
	m_mri_size[0] = m_parameters->mri_size[0];
	m_mri_size[1] = m_parameters->mri_size[1];
	m_mri_size[2] = m_parameters->mri_size[2];
numberOfPixels_MRI = m_mri_size[0] * m_mri_size[1] * m_mri_size[2];

	m_mri_spacing[0] = m_parameters->mri_spacing[0];
	m_mri_spacing[1] = m_parameters->mri_spacing[1];
	m_mri_spacing[2] = m_parameters->mri_spacing[2];

	m_mri_origen[0] = m_parameters->mri_origen[0];
	m_mri_origen[1] = m_parameters->mri_origen[1];
	m_mri_origen[2] = m_parameters->mri_origen[2];

	m_mri_imagepointer = new float[ numberOfPixels_MRI ];
		for(int i=0; i<numberOfPixels_MRI; i++) m_mri_imagepointer[i] = m_parameters->mri_imagePointer[i];

numberOfPoints = m_parameters->numberOfPoints;
	m_i_points = new float[ 3*numberOfPoints ];
		for( int i=0; i<3*numberOfPoints; i++) m_i_points[i] = m_parameters->initial_points[i];
	m_f_points = new float[ 3*numberOfPoints ];
		for(int i=0; i<3*numberOfPoints; i++) m_f_points[i] = m_parameters->final_points[i];
numberOfElements = m_parameters->numberOfElements;
	m_elements = new int[4*numberOfElements];	
	for(int i=0; i<4*numberOfElements; i++) m_elements[i] = m_parameters->elements[i];

	m_grid_origen[0] = m_parameters->grid_origen[0];
	m_grid_origen[1] = m_parameters->grid_origen[1];
	m_grid_origen[2] = m_parameters->grid_origen[2];

	m_grid_spacing[0] = m_parameters->grid_spacing[0];
	m_grid_spacing[1] = m_parameters->grid_spacing[1];
	m_grid_spacing[2] = m_parameters->grid_spacing[2];

	m_grid_size[0] = m_parameters->grid_size[0];
	m_grid_size[1] = m_parameters->grid_size[1];
	m_grid_size[2] = m_parameters->grid_size[2];
numberOfVoxelsGrid = m_grid_size[0] * m_grid_size[1] * m_grid_size[2] ; 

	m_flags = new int[ numberOfVoxelsGrid ];
		for(int i=0; i<numberOfVoxelsGrid; i++) m_flags[i] = m_parameters->flags[i];
	m_cumsum = new int[ numberOfVoxelsGrid ];
		for(int i=0; i<numberOfVoxelsGrid; i++) m_cumsum[i] = m_parameters->cumsum[i];
maximumCorrespondingElements = m_cumsum[ numberOfVoxelsGrid -1];
	m_correspondingElements = new int[ maximumCorrespondingElements];
	for (int i=0; i< maximumCorrespondingElements; i++)	m_correspondingElements[i] = m_parameters->correspondingElements[i];

	m_simulada_size[0] = m_parameters->mamo_size[0];
	m_simulada_size[1] = m_parameters->mamo_size[1];
	m_simulada_size[2] = m_parameters->mamo_size[2];
numberOfPixels_Simulada = m_simulada_size[0] * m_simulada_size[1];

	m_simulada_origen[0] = m_parameters->mamo_origen[0];
	m_simulada_origen[1] = m_parameters->mamo_origen[1];
	m_simulada_origen[2] = m_parameters->mamo_origen[2];

	m_simulada_spacing[0] = m_parameters->mamo_spacing[0];
	m_simulada_spacing[1] = m_parameters->mamo_spacing[1];
	m_simulada_spacing[2] = m_parameters->mamo_spacing[2];
	
	//m_simulada_imagepointer = new float[numberOfPixels_Simulada];
	m_simulada_imagepointer = new unsigned short[numberOfPixels_Simulada];

	m_source[0] = m_parameters->source[0];
	m_source[1] = m_parameters->source[1];
	m_source[2] = m_parameters->source[2];
}

// Metodos
void CudaProjection::Update()
{
	/*
	printf("\n");
	printf("Entra en cuda\n");
	printf("\n");

// Timer !
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);
	*/

	Initialize();

// Allocacion de la memoria GPU !
	// MRI !
    cudaStatus = hipMalloc((void**)&dev_mri_size, 3*sizeof(int));
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 3d Size!\n");   // MRI Size
	cudaStatus = hipMalloc((void**)&dev_mri_spacing, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 3d Spacing!\n"); // MRI Spacing
	cudaStatus = hipMalloc((void**)&dev_mri_origen, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 3d Origen!\n");  // MRI Origen

	cudaStatus = hipMalloc((void**)&dev_mri_imagepointer, numberOfPixels_MRI*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 3d image pointer!\n");  // MRI imagen !!

	// Mesh & Grid !!
	cudaStatus = hipMalloc((void**)&dev_i_points, 3*numberOfPoints*sizeof(float)); 
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc i_point !\n"); // i_points
	cudaStatus = hipMalloc((void**)&dev_f_points, 3*numberOfPoints*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc f_point !\n"); // f_points

	cudaStatus = hipMalloc((void**)&dev_elements, 4*numberOfElements*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc elements !\n"); // elements;

	cudaStatus = hipMalloc((void**)&dev_grid_origen, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Grid Origen !\n"); // grid_origen
	cudaStatus = hipMalloc((void**)&dev_grid_spacing, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Grid spacing!\n"); // grid_Spacing
	cudaStatus = hipMalloc((void**)&dev_grid_size, 3*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Grid Size !\n"); // grid_Size

	cudaStatus = hipMalloc((void**)&dev_flags, numberOfVoxelsGrid*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc Flags !\n"); // flags
	cudaStatus = hipMalloc((void**)&dev_cumsum, numberOfVoxelsGrid*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc CumSum !\n"); // cumsum
	cudaStatus = hipMalloc((void**)&dev_correspondingElements, maximumCorrespondingElements*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA alloc corresponding element !\n"); // corresponding Elements !

	// Imagen Simulada !
	cudaStatus = hipMalloc((void**)&dev_simulada_size, 3*sizeof(int));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 2d Size!\n");  // Simulada Size
	cudaStatus = hipMalloc((void**)&dev_simulada_spacing, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 2d Spacing!\n");  // Simulada Spacing
	cudaStatus = hipMalloc((void**)&dev_simulada_origen, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 2d Origen!\n");  // simulada Origen

	cudaStatus = hipMalloc((void**)&dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(unsigned short));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 2d image pointer!\n");  // Simulada Imagen !!
	cudaStatus = hipMemset((void*)dev_simulada_imagepointer, 0, numberOfPixels_Simulada*sizeof(unsigned short));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc 2d image pointer!\n");  // Inicializaci�n de la imagen simulada a Zeros !!

	// Source !
	cudaStatus = hipMalloc((void**)&dev_source, 3*sizeof(float));
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA alloc Source!\n");  // source !"


// Copia a memoria device
	// mri
	cudaStatus = hipMemcpy(dev_mri_size, (const int*) m_mri_size, 3*sizeof(int), hipMemcpyHostToDevice);
	// cudaStatus = hipMemcpy(dev_mri_size, (const int*) m_mri_size, 3*sizeof(int), hipMemcpyHostToDevice);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Size Host2Dev!\n");
	cudaStatus = hipMemcpy(dev_mri_spacing, (const float*) m_mri_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
	// cudaStatus = hipMemcpy(dev_mri_spacing, this->m_mri_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Spacing Host2Dev!\n");
	cudaStatus = hipMemcpy(dev_mri_origen, (const float*) m_mri_origen, 3*sizeof(float), hipMemcpyHostToDevice);
	// cudaStatus = hipMemcpy(dev_mri_origen, this->m_mri_origen, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Origen Host2Dev!\n");
	cudaStatus = hipMemcpy(dev_mri_imagepointer, (const float*) m_mri_imagepointer, numberOfPixels_MRI*sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_mri_imagepointer, this->m_mri_imagepointer, numberOfPixels_MRI*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d image pointer Host2Dev!\n");

	// Mesh & Grid !!
	cudaStatus = hipMemcpy(dev_i_points, (const float*) m_i_points, 3*numberOfPoints*sizeof(float), hipMemcpyHostToDevice); 
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy i_point Host2Dev !\n"); // i_points
	cudaStatus = hipMemcpy(dev_f_points, (const float*) m_f_points,  3*numberOfPoints*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy f_point Host2Dev !\n"); // f_points

	cudaStatus = hipMemcpy(dev_elements, (const int*) m_elements, 4*numberOfElements*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy elements Host2Dev !\n"); // elements;

	cudaStatus = hipMemcpy(dev_grid_origen, (const float*) m_grid_origen, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy Grid Origen !\n"); // grid_origen
	cudaStatus = hipMemcpy(dev_grid_spacing, (const float*) m_grid_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy Grid spacing!\n"); // grid_Spacing
	cudaStatus = hipMemcpy(dev_grid_size, (const int*) m_grid_size, 3*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcopy Grid Size !\n"); // grid_Size

	cudaStatus = hipMemcpy(dev_flags, (const int*) m_flags, numberOfVoxelsGrid*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcpy Flags !\n"); // flags
	cudaStatus = hipMemcpy(dev_cumsum, (const int*) m_cumsum, numberOfVoxelsGrid*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcpy CumSum !\n"); // cumsum
	cudaStatus = hipMemcpy(dev_correspondingElements, (const int*) m_correspondingElements, maximumCorrespondingElements*sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf( stderr, "CUDA memcpy corresponding element !\n"); // corresponding Elements !

	// mamo simulada
	cudaStatus = hipMemcpy(dev_simulada_size, (const int*) m_simulada_size, 3*sizeof(int), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_simulada_size, this->m_simulada_size, 3*sizeof(int), hipMemcpyHostToDevice);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d Size Host2Dev!\n");
	cudaStatus = hipMemcpy(dev_simulada_spacing, (const float*) m_simulada_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_simulada_spacing, this->m_simulada_spacing, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d Spacing Host2Dev!\n");
	cudaStatus = hipMemcpy(dev_simulada_origen, (const float*) m_simulada_origen, 3*sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_simulada_origen, this->m_simulada_origen, 3*sizeof(float), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d Origen Host2Dev!\n");
//	cudaStatus = hipMemcpy(dev_2d_imagepointer, this->m_2d_imagepointer, numberOfPixels_2D*sizeof(float), hipMemcpyHostToDevice);
//		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d image pointer Host2Dev!\n");

	cudaStatus = hipMemcpy(dev_source, (const float*) m_source, 3*sizeof(float), hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy(dev_source, this->m_source, 3*sizeof(float), hipMemcpyHostToDevice);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Source Host2Dev!\n");

/*
		// Checkin memory !!
		printf("\n");
		printf("3d size: [ %i,%i,%i]\n", m_3d_size[0],m_3d_size[1],m_3d_size[2]);
		printf("3D Spacing : [ %f,%f,%f]\n",m_3d_spacing[0], m_3d_spacing[1], m_3d_spacing[2]);
		printf("3D Origen : [ %f,%f,%f]\n",m_3d_origen[0], m_3d_origen[1], m_3d_origen[2]);
		
		printf("2D Size : [ %d,%d,%d]\n",m_2d_size[0], m_2d_size[1], m_2d_size[2]);
		printf("2D Spacing : [ %f,%f,%f]\n",m_2d_spacing[0], m_2d_spacing[1], m_2d_spacing[2]);
		printf("2D Origen : [ %f,%f,%f]\n",m_2d_origen[0], m_2d_origen[1], m_2d_origen[2]);
		
		printf("Source : [ %f,%f,%f]\n",m_source[0], m_source[1], m_source[2]);
		printf("\n");
*/

		bl = (int)ceilf((float)(numberOfPixels_Simulada/128))+1;
// Kernel de proyecci�n?
//	 fill_dos <<< bl,128 >>> (dev_simulada_imagepointer);

//	printf("Entra en el kernel\n" );

	//cudaStatus = hipSetDevice(0);
	kernel_projection <<< bl,128 >>> (dev_mri_size, dev_mri_spacing, dev_mri_origen, dev_mri_imagepointer,
										dev_i_points, dev_f_points, dev_elements,
										dev_grid_origen, dev_grid_spacing, dev_grid_size,
										dev_flags, dev_cumsum, dev_correspondingElements,
									  dev_simulada_size, dev_simulada_spacing, dev_simulada_origen, dev_simulada_imagepointer,
									  dev_source);								   
//	printf("Sale del kernel\n" );							   
	hipDeviceSynchronize();
// Copia a memoria host
/*	int temp_3dsize[3] = {0,0,0};
	cudaStatus = hipMemcpy( temp_3dsize, (const int*) dev_3d_size, 3*sizeof(int),  hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Size Dev2Host!\n");
		else m_3d_size = temp_3dsize;

	float temp_3dspacing[3] = {0.0, 0.0,0.0};
	cudaStatus = hipMemcpy( temp_3dspacing, (const float*) dev_3d_spacing, 3*sizeof(float),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Spacing Dev2Host!\n");
		else m_3d_spacing = temp_3dspacing;

	float temp_3dorigen[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_3dorigen, (const float*) dev_3d_origen, 3*sizeof(float),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d Origen Dev2Host!\n");
		else m_3d_origen = temp_3dorigen;

	float* image3dpointer = new float[numberOfPixels_3D];
	cudaStatus = hipMemcpy( image3dpointer, (const float*) dev_3d_imagepointer, numberOfPixels_3D*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 3d image pointer Dev2Host!\n");
		else m_3d_imagepointer = image3dpointer;

	int temp_2dsize[3] = {0,0,0};
	cudaStatus = hipMemcpy( temp_2dsize, (const int*) dev_2d_size, 3*sizeof(int), hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d Size Dev2Host!\n");
		else m_2d_size = temp_2dsize;

	float temp_2dspacing[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_2dspacing, (const float*) dev_2d_spacing, 3*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d Spacing Dev2Host!\n");
		else m_2d_spacing = temp_2dspacing;

	float temp_2dorigen[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_2dorigen, (const float*) dev_2d_origen, 3*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d Origen Dev2Host!\n");
		else m_2d_origen = temp_2dorigen;
*/
//	printf("va a inicializar con pixels \n" );
	//float * temp_image2dpointer;
	 //float * temp_image2dpointer = new float[numberOfPixels_Simulada]; // El m_... no convenci� quiz� porque no esta ba inicializado...

	//cudaStatus = hipMemcpy(temp_image2dpointer, (const float*) dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(float),  hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(m_simulada_imagepointer, (const unsigned short*)dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(unsigned short),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d image pointer Dev2Host!\n");
		else  m_parameters->simulada_imagePointer = m_simulada_imagepointer;
		

/*  AQUI VA LA RECUPERACION DE LA IMAGEN ORIGINAL !! RECUERDALO PORQUE ESTO HABRA QUE CAMBIARLO

	printf("va a inicializar con pixels \n" );
	//float * temp_image2dpointer;
	 float * temp_image2dpointer = new float[numberOfPixels_Simulada]; // El m_... no convenci� quiz� porque no esta ba inicializado...
	//cudaStatus = hipMemcpy(temp_image2dpointer, (const float*) dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(float),  hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(temp_image2dpointer, (const float*)dev_simulada_imagepointer, numberOfPixels_Simulada*sizeof(float),  hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy 2d image pointer Dev2Host!\n");
		else {
			m_simulada_imagepointer = temp_image2dpointer;
			m_parameters->simulada_imagePointer = temp_image2dpointer;
		}


*/


/*	float temp_source[3] = {0.0,0.0,0.0};
	cudaStatus = hipMemcpy( temp_source, (const float*) dev_source, 3*sizeof(float), hipMemcpyDeviceToHost);
	    if (cudaStatus != hipSuccess) fprintf(stderr, "CUDA memcopy Source Dev2Host!\n");
		else m_source = temp_source;
*/

/*
	printf("\n");
	printf("3d size: [ %i,%i,%i]\n", m_3d_size[0],m_3d_size[1],m_3d_size[2]);
	printf("3D Spacing : [ %f,%f,%f]\n",m_3d_spacing[0], m_3d_spacing[1], m_3d_spacing[2]);
	printf("3D Origen : [ %f,%f,%f]\n",m_3d_origen[0], m_3d_origen[1], m_3d_origen[2]);
	printf("3D Image Pointer : [ %f,%f,%f, ...]\n",m_3d_imagepointer[0], m_3d_imagepointer[1], m_3d_imagepointer[2]);


	printf("2D Size : [ %d,%d,%d]\n",m_2d_size[0], m_2d_size[1], m_2d_size[2]);
	printf("2D Spacing : [ %f,%f,%f]\n",m_2d_spacing[0], m_2d_spacing[1], m_2d_spacing[2]);
	printf("2D Origen : [ %f,%f,%f]\n",m_2d_origen[0], m_2d_origen[1], m_2d_origen[2]);
	printf("2D Image Pointer : [ %f,%f,%f, ...]\n",m_2d_imagepointer[0], m_2d_imagepointer[1], m_2d_imagepointer[2]);

	printf("Source : [ %f,%f,%f]\n",m_source[0], m_source[1], m_source[2]);
	printf("\n");
*/

// Liberando memoria !!
	hipFree( (void*) dev_mri_size);				//hipFree( temp_3dsize);
	hipFree( (void*) dev_mri_spacing);			//hipFree( temp_3dspacing);
	hipFree( (void*) dev_mri_origen );			//hipFree( temp_3dorigen);
	hipFree( (void*) dev_mri_imagepointer );	//hipFree( image3dpointer);

	hipFree( (void*) dev_i_points );
	hipFree( (void*) dev_f_points );
	hipFree( (void*) dev_elements );

	hipFree( (void*) dev_grid_origen );
	hipFree( (void*) dev_grid_spacing );
	hipFree( (void*) dev_grid_size );

	hipFree( (void*) dev_flags );
	hipFree( (void*) dev_cumsum );
	hipFree( (void*) dev_correspondingElements );

	hipFree( (void*) dev_simulada_size);				//hipFree( temp_2dsize);
	hipFree( (void*) dev_simulada_spacing);			//hipFree( temp_2dspacing);
	hipFree( (void*) dev_simulada_origen);			//hipFree( temp_2dorigen);
	hipFree( (void*) dev_simulada_imagepointer);		//hipFree( (void*) temp_image2dpointer);

	//hipFree( numberOfPixels_3D);		hipFree( numberOfPixels_2D);

	hipFree( (void*) dev_source);				//hipFree( temp_source);
	hipDeviceReset();

	hipFree( kernel_projection );

// Time !!
/*	hipEventRecord(stop,0);
	//hipEventSynchronize( stop);
	hipEventElapsedTime( &time, start, stop);
	printf( "Time: %f ms.\n", time);

	printf("\n");
	printf("Sale de cuda\n");
	printf("\n");
	*/
}

