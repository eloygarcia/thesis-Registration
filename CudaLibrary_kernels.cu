#include "hip/hip_runtime.h"



__device__ void computealpha( const int  ind, const float origen, const float voxelSize,const float p1, const float p2, float &alpha) {	// alpha = [ X_plane(i) - X1] / [X2-X1] ;
		alpha = ((ind*voxelSize + origen)-p1)/(p2-p1); };

__device__ void Determinante( float* v0, float* v1, float* v2, float* v3, float & det )
{
	//float det = 0.0;
	det  = (v0[2]*v1[1]*v2[0]*1 - v0[1]*v1[2]*v2[0]*1 - 
			v0[2]*v1[0]*v2[1]*1 + v0[0]*v1[2]*v2[1]*1 + 
			v0[1]*v1[0]*v2[2]*1 - v0[0]*v1[1]*v2[2]*1 - 
			v0[2]*v1[1]*1*v3[0] + v0[1]*v1[2]*1*v3[0] + 
			v0[2]*1*v2[1]*v3[0] - 1*v1[2]*v2[1]*v3[0] - 
			v0[1]*1*v2[2]*v3[0] + 1*v1[1]*v2[2]*v3[0] + 
			v0[2]*v1[0]*1*v3[1] - v0[0]*v1[2]*1*v3[1] - 
			v0[2]*1*v2[0]*v3[1] + 1*v1[2]*v2[0]*v3[1] + 
			v0[0]*1*v2[2]*v3[1] - 1*v1[0]*v2[2]*v3[1] - 
			v0[1]*v1[0]*1*v3[2] + v0[0]*v1[1]*1*v3[2] + 
			v0[1]*1*v2[0]*v3[2] - 1*v1[1]*v2[0]*v3[2] - 
			v0[0]*1*v2[1]*v3[2] + 1*v1[0]*v2[1]*v3[2])/6;

	// return det;
}

__device__ void triLinearInterpolator(const float* imagepointer, const int* size, const float* pixel, float & value) // es un pixel continuo, con lo cual, vale
{
//	float value = 0.0;
	int x0, y0, z0, x1, y1, z1;
		x0 = (int) (floor(pixel[0])); x1=x0+1; // x1 = (int) (ceilf(pixel[0]));
		y0 = (int) (floor(pixel[1])); y1=y0+1; // y1 = (int) (ceilf(pixel[1]));
		z0 = (int) (floor(pixel[2])); z1=z0+1; // z1 = (int) (ceilf(pixel[2]));

	float xd = (pixel[0]-x0)/(x1-x0);
	float yd = (pixel[1]-y0)/(y1-y0);
	float zd = (pixel[2]-z0)/(z1-z0);

	int idx_1 = 0;		int idx_2 = 0;

		idx_1 = x0 + (size[0] * y0) + (size[0]*size[1]*z0);
		idx_2 = x1 + (size[0] * y0) + (size[0]*size[1]*z0);
	float c00 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );
		idx_1 = x0 + (size[0] * y0) + (size[0]*size[1]*z1);
		idx_2 = x1 + (size[0] * y0) + (size[0]*size[1]*z1);
	float c01 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );
		idx_1 = x0 + (size[0] * y1) + (size[0]*size[1]*z0);
		idx_2 = x1 + (size[0] * y1) + (size[0]*size[1]*z0);
	float c10 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );
		idx_1 = x0 + (size[0] * y1) + (size[0]*size[1]*z1);
		idx_2 = x1 + (size[0] * y1) + (size[0]*size[1]*z1);
	float c11 = ( imagepointer[idx_1] * (1-xd) ) + ( imagepointer[ idx_2 ] * (xd) );

		float c0 = c00 * (1-yd) + c10 * yd;
		float c1 = c01 * (1-yd) + c11 * yd;

		float c = c0 * (1-zd) + c1 * zd;

		value = c;
}

__device__ void computeBaricentricCoordinates(float* posicionPunto, float* vertex_0, float* vertex_1, float* vertex_2, float* vertex_3, float* baricentricCoordinates, bool & is_inside)
{
	baricentricCoordinates[0] = 0.0;
	baricentricCoordinates[1] = 0.0;
	baricentricCoordinates[2] = 0.0;
	baricentricCoordinates[3] = 0.0;

	//float V = 0;
	float V = 0;
	Determinante( vertex_0, vertex_1, vertex_2, vertex_3, V );

	//float v1 = 0; 
	float v1 = 0;
	Determinante( posicionPunto, vertex_1, vertex_2, vertex_3, v1);
	//float v2 = 0;
	float v2 = 0;
	Determinante( vertex_0, posicionPunto, vertex_2, vertex_3, v2);
	//float v3 = 0;
	float v3 = 0;
	Determinante( vertex_0, vertex_1, posicionPunto, vertex_3, v3);
	//float v4 = 0;
	float v4 = 0;
	Determinante( vertex_0, vertex_1, vertex_2 ,posicionPunto, v4);
		
	//bool is_inside = false;
	is_inside = false;

	if( ((v1/V)>= -.01 && (v1/V)<= 1.01) && ((v2/V)>= -0.01 && (v2/V)<= 1.01) && ((v3/V)>= -0.01 && (v3/V)<=1.01) && ((v4/V)>= -0.01 && (v4/V)<= 1.01) )
	//if( ((v1/V)>=0 && (v1/V)<= 1) && ((v2/V)>=0 && (v2/V)<= 1) && ((v3/V)>=0 && (v3/V)<=1) && ((v4/V)>=0 && (v4/V)<=1) )
	{
		/*
		std::cout << std::endl;
		std::cout << " punto : [" << posicionPunto[0] << ", " << posicionPunto[1] << ", " << posicionPunto[2] << "] " << std::endl;
		std::cout << "vertex_0 : [" << vertex_0[0] << ", " << vertex_0[1] << ", " << vertex_0[2] << "] " << std::endl;
		std::cout << "vertex_1 : [" << vertex_1[0] << ", " << vertex_1[1] << ", " << vertex_1[2] << "] " << std::endl;
		std::cout << "vertex_2 : [" << vertex_2[0] << ", " << vertex_2[1] << ", " << vertex_2[2] << "] " << std::endl;
		std::cout << "vertex_3 : [" << vertex_3[0] << ", " << vertex_3[1] << ", " << vertex_3[2] << "] " << std::endl;
		std::cout << std::endl;

		std::cout << "Baricentric Coordinates : [" << v1/V <<", " << v2/V << ", " << v3/V << ", " << v4/V << "] " << std::endl;
		*/
		is_inside = true;
		/**/
		baricentricCoordinates[0] = v1/V; 
		baricentricCoordinates[1] = v2/V;
		baricentricCoordinates[2] = v3/V;
		baricentricCoordinates[3] = v4/V;

	}

	// std::cout << "Baricentric Coordinates : [" << v1/V <<", " << v2/V << ", " << v3/V << ", " << v4/V << "] " << std::endl;

	// Sleep(2000);

	// return is_inside;
}

__device__ void computeCartessianCoordinates(float* baricentricCoordinates, float* vertex_0, float* vertex_1, float* vertex_2, float* vertex_3, float* posicionPunto)
{
	posicionPunto[0] = baricentricCoordinates[0]*vertex_0[0] + baricentricCoordinates[1]*vertex_1[0] + baricentricCoordinates[2]*vertex_2[0] + baricentricCoordinates[3]*vertex_3[0];
	posicionPunto[1] = baricentricCoordinates[0]*vertex_0[1] + baricentricCoordinates[1]*vertex_1[1] + baricentricCoordinates[2]*vertex_2[1] + baricentricCoordinates[3]*vertex_3[1];
	posicionPunto[2] = baricentricCoordinates[0]*vertex_0[2] + baricentricCoordinates[1]*vertex_1[2] + baricentricCoordinates[2]*vertex_2[2] + baricentricCoordinates[3]*vertex_3[2];
}

/*
__global__ void kernel_projection(const int* dev_3d_size, const float* dev_3d_spacing, const float* dev_3d_origen, const float* dev_3d_imagepointer,
								  const int* dev_2d_size, const float* dev_2d_spacing, const float* dev_2d_origen, float* dev_2d_imagepointer,
								  const float* source)
{
	int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	int numberOfPixels2d = dev_2d_size[0] * dev_2d_size[1]; // *size2d[2];

	if( i<numberOfPixels2d){

	// Fila y columna de la imagen !
	int row = (int) floorf( i / dev_2d_size[0] ); 
	int col = (int) ( i - (row*dev_2d_size[0]) ); 
	

	// Posici�n del pixel del detector !!
	float x2 = dev_2d_origen[0] + (col * dev_2d_spacing[0] );
	float y2 = dev_2d_origen[1] + (row * dev_2d_spacing[1] );

	// Vector de direcci�n !!
	float vect[3];
		vect[0] = x2 - source[0];
		vect[1] = y2 - source[1];
		vect[2] = dev_2d_origen[2] - source[2];  // Revisar este punto.

	// Distancia de la fuente al detector !!
	float xa = pow(vect[0],2);
	float ya = pow(vect[1],2);
	float za = pow(vect[2],2);

	float dist12 = sqrt( xa + ya + za );

	// C�lculo del alpha en Z !!
	float temp = 0.0f;
	computealpha( 0, dev_3d_origen[2], dev_3d_spacing[2], source[2], dev_2d_origen[2], temp );

	// Resolver la ecuaci�n de la recta !!
	float temp_dist[3] = {0.0f, 0.0f, 0.0f};
	float point[3] = {0.0f, 0.0f, 0.0f};
	float step = 0.0005f;
	float t=temp;

	float pixel[3] = {0.0f, 0.0f, 0.0f};

	float value = 0.0f;
	float length = 0.0f;

	// Longitud del paso !!
	temp_dist[0] = pow(step*vect[0],2);
	temp_dist[1] = pow(step*vect[1],2);
	temp_dist[2] = pow(step*vect[2],2);
	float l_step = sqrt(temp_dist[0] + temp_dist[1] + temp_dist[2]);

	while( t<1 )
	{
		// Calculo del siguiente punto en la recta
		point[0] = source[0] + t * vect[0];
		point[1] = source[1] + t * vect[1];
		point[2] = source[2] + t * vect[2];
		// Posici�n del voxel parcial !!
		pixel[0] = (point[0] - dev_3d_origen[0]) / dev_3d_spacing[0];
		pixel[1] = (point[1] - dev_3d_origen[1]) / dev_3d_spacing[1];
		pixel[2] = (point[2] - dev_3d_origen[2]) / dev_3d_spacing[2];
		// Interpolaci�n trilienal
		if((pixel[0]<0 || pixel[0]>dev_3d_size[0]-1) || (pixel[1]<0 || pixel[1]>dev_3d_size[1]-1) || (pixel[2]<0 || pixel[2]>dev_3d_size[2]-1))  value = 0;
		else triLinearInterpolator( dev_3d_imagepointer, dev_3d_size, pixel, value);
		// Longitud acumulada !!
		length += (1000 * value * l_step);
		// Nuevo punto!! 
		t+=step;
	}

	if(length>0 & length < 65535 ) dev_2d_imagepointer[ i ] = length;
	else dev_2d_imagepointer[ i ] = 0.0f;
	
	}
}
*/
__global__ void kernel_projection(const int* dev_3d_size, const float* dev_3d_spacing, const float* dev_3d_origen, const float* dev_3d_imagepointer,
								  const float* dev_i_points, const float* dev_f_points, const int* dev_elements,
								  const float* dev_grid_origen, const float* dev_grid_spacing, const int* dev_grid_size,
								  const int* dev_flags, const int* dev_cumsum, const int* dev_correspondingElements,
								  const int* dev_2d_size, const float* dev_2d_spacing, const float* dev_2d_origen, unsigned short* dev_2d_imagepointer, //float* dev_2d_imagepointer,
								  const float* dev_source)
{
	int i = (blockDim.x * blockIdx.x) + threadIdx.x;
	
	int numberOfPixels2d = dev_2d_size[0] * dev_2d_size[1]; // *size2d[2];

//	if( i<numberOfPixels2d){
		// Fila y columna de la imagen !
		int row = (int) floorf( i / dev_2d_size[0] ); 
		int col = (int) ( i - (row*dev_2d_size[0]) ); 

		// Posici�n del pixel del detector !!
		float x2 = dev_2d_origen[0] + (col * dev_2d_spacing[0] );
		float y2 = dev_2d_origen[1] + (row * dev_2d_spacing[1] );

		// Vector de direcci�n !!
		float vect[3];
			vect[0] = x2 - dev_source[0];
			vect[1] = y2 - dev_source[1];
			vect[2] = dev_2d_origen[2] - dev_source[2];  // Revisar este punto.

		// Distancia de la fuente al detector !!
		float xa = pow(vect[0],2);
		float ya = pow(vect[1],2);
		float za = pow(vect[2],2);

		float dist12 = sqrt( xa + ya + za );

		// C�lculo del alpha en Z !!
		float temp = 0.0f;
		// En este caso el alpha se calcula en relaci�n a la grid !!
		computealpha( 0, dev_grid_origen[2], dev_grid_spacing[2], dev_source[2], dev_2d_origen[2], temp ); 

		// Resolver la ecuaci�n de la recta !!
		float step = 0.0005f;
		float t=temp;
		// on the grid !
		float temp_dist[3] = {0.0f, 0.0f, 0.0f};  // temporal distance
		float point[3] = {0.0f, 0.0f, 0.0f};  // physical point on the grid
		
		int voxel_grid[3] = {0,0,0};
		float baricentricCoordinates[4] = {0.0f, 0.0f, 0.0f, 0.0f};

		int index = 0;

		int index_start = 0;
		int number_of_elements_here = 0;
		// int count = 0;

		int element_index_number = 0;
		int element_number = 0;

		int numberofVoxelsGrid = dev_grid_size[0] * dev_grid_size[1] * dev_grid_size[2];

		float vertex_0[3] = {0.0f,0.0f,0.0f};
		float vertex_1[3] = {0.0f,0.0f,0.0f};
		float vertex_2[3] = {0.0f,0.0f,0.0f};
		float vertex_3[3] = {0.0f,0.0f,0.0f};

		bool is_inside = false;

		//float old_vertex_0[3] = {0.0f,0.0f,0.0f};
		//float old_vertex_1[3] = {0.0f,0.0f,0.0f};
		//float old_vertex_2[3] = {0.0f,0.0f,0.0f};
		//float old_vertex_3[3] = {0.0f,0.0f,0.0f};

		// on the mri
		float position_mri[3] = {0.0f, 0.0f, 0.0f};
		float pixel_mri[3] = {0.0f, 0.0f, 0.0f}; // el pixel es el voxel de a grid !! // No lo necesito?'
		
		// Solving the step !
		float value = 0.0f;
		float length = 0.0f;

		// Longitud del paso !!
		temp_dist[0] = pow(step*vect[0],2);
		temp_dist[1] = pow(step*vect[1],2);
		temp_dist[2] = pow(step*vect[2],2);
		float l_step = sqrt(temp_dist[0] + temp_dist[1] + temp_dist[2]);

		int max_num_of_elem_here = 0;

		while( t<1 )
		{
			// Calculo del siguiente punto en la recta
			point[0] = dev_source[0] + t * vect[0];
			point[1] = dev_source[1] + t * vect[1];
			point[2] = dev_source[2] + t * vect[2];
			
			// Voxel on the grid //
			voxel_grid[0] = (int)(floorf((point[0] - dev_grid_origen[0])/dev_grid_spacing[0]));
			voxel_grid[1] = (int)(floorf((point[1] - dev_grid_origen[1])/dev_grid_spacing[1]));
			voxel_grid[2] = (int)(floorf((point[2] - dev_grid_origen[2])/dev_grid_spacing[2]));

			if( ((voxel_grid[0]>0) && (voxel_grid[0]<dev_grid_size[0]) ) &&
				((voxel_grid[1]>0) && (voxel_grid[1]<dev_grid_size[1]) ) &&
				((voxel_grid[2]>0) && (voxel_grid[2]<dev_grid_size[2]) ) )
			{
				index =(int)( (voxel_grid[2]*(dev_grid_size[0]*dev_grid_size[1])) + (voxel_grid[1]*dev_grid_size[0]) + voxel_grid[0]);
				
				index_start = (int)dev_cumsum[ index ];
				number_of_elements_here = (int) dev_flags[ index ];
				
				if(number_of_elements_here!=0){
					for( int j=0; j<number_of_elements_here; j++){
						element_index_number = index_start + j;
						if( element_index_number < dev_cumsum[ numberofVoxelsGrid -1] ){
							element_number =  dev_correspondingElements[ element_index_number ];

							// float temp_vertex_0[3] = {0.0,0.0,0.0};
							vertex_0[0] = dev_f_points[ 3*dev_elements[ 4*element_number ]  ];
							vertex_0[1] = dev_f_points[ 3*dev_elements[ 4*element_number ] +1 ];
							vertex_0[2] = dev_f_points[ 3*dev_elements[ 4*element_number ] +2 ];
					
							// float temp_vertex_1[3] = {0.0,0.0,0.0};
							vertex_1[0] = dev_f_points[ 3*dev_elements[ 4*element_number +1 ] ];
							vertex_1[1] = dev_f_points[ 3*dev_elements[ 4*element_number +1 ] +1 ];
							vertex_1[2] = dev_f_points[ 3*dev_elements[ 4*element_number +1 ] +2 ];
				
							// float temp_vertex_2[3] = {0.0,0.0,0.0};
							vertex_2[0] = dev_f_points[ 3*dev_elements[ 4*element_number +2 ] ];
							vertex_2[1] = dev_f_points[ 3*dev_elements[ 4*element_number +2 ] +1 ];
							vertex_2[2] = dev_f_points[ 3*dev_elements[ 4*element_number +2 ] +2 ];
				
							// float temp_vertex_3[3] = {0.0,0.0,0.0};
							vertex_3[0] = dev_f_points[ 3*dev_elements[ 4*element_number +3 ] ];
							vertex_3[1] = dev_f_points[ 3*dev_elements[ 4*element_number +3 ] +1 ];
							vertex_3[2] = dev_f_points[ 3*dev_elements[ 4*element_number +3 ] +2 ];

							computeBaricentricCoordinates(point, vertex_0, vertex_1, vertex_2, vertex_3, baricentricCoordinates, is_inside);
				
							if( is_inside ){
								vertex_0[0] = dev_i_points[ 3*dev_elements[ 4*element_number ]  ];
								vertex_0[1] = dev_i_points[ 3*dev_elements[ 4*element_number ] +1 ];
								vertex_0[2] = dev_i_points[ 3*dev_elements[ 4*element_number ] +2 ];
					
								// float temp_vertex_1[3] = {0.0,0.0,0.0};
								vertex_1[0] = dev_i_points[ 3*dev_elements[ 4*element_number +1 ] ];
								vertex_1[1] = dev_i_points[ 3*dev_elements[ 4*element_number +1 ] +1 ];
								vertex_1[2] = dev_i_points[ 3*dev_elements[ 4*element_number +1 ] +2 ];
					
								// float temp_vertex_2[3] = {0.0,0.0,0.0};
								vertex_2[0] = dev_i_points[ 3*dev_elements[ 4*element_number +2 ] ];
								vertex_2[1] = dev_i_points[ 3*dev_elements[ 4*element_number +2 ] +1 ];
								vertex_2[2] = dev_i_points[ 3*dev_elements[ 4*element_number +2 ] +2 ];
				
								// float temp_vertex_3[3] = {0.0,0.0,0.0};
								vertex_3[0] = dev_i_points[ 3*dev_elements[ 4*element_number +3 ] ];
								vertex_3[1] = dev_i_points[ 3*dev_elements[ 4*element_number +3 ] +1 ];
								vertex_3[2] = dev_i_points[ 3*dev_elements[ 4*element_number +3 ] +2 ];
		
								computeCartessianCoordinates( baricentricCoordinates, vertex_0, vertex_1, vertex_2, vertex_3, position_mri);

								// Posici�n del voxel parcial !! // esto tiene que ser on the grid
								pixel_mri[0] = (float)((position_mri[0] - dev_3d_origen[0]) / dev_3d_spacing[0]);
								pixel_mri[1] = (float)((position_mri[1] - dev_3d_origen[1]) / dev_3d_spacing[1]);
								pixel_mri[2] = (float)((position_mri[2] - dev_3d_origen[2]) / dev_3d_spacing[2]);
									
								// Interpolaci�n trilineal
								if(((pixel_mri[0]>0.0f) && (pixel_mri[0]<(float)(dev_3d_size[0]))) &&
								   ((pixel_mri[1]>0.0f) && (pixel_mri[1]<(float)(dev_3d_size[1]))) &&
								   ((pixel_mri[2]>0.0f) && (pixel_mri[2]<(float)(dev_3d_size[2])))) 
									{ 
									//	value = 1.0f;
										triLinearInterpolator( dev_3d_imagepointer, dev_3d_size, pixel_mri, value);
										//max_num_of_elem_here = 0.0f;
								} else {
									value = 0.0f;
									//max_num_of_elem_here = 1.0f;
								}
	
								// Longitud acumulada !!
								length += (1000 * value * l_step);
								break;
							}
						}
					}
				}
			}
			// Nuevo Punto !!
			t+=step;
		}

		if(length>0 & length < 65535 ) dev_2d_imagepointer[ i ] = (unsigned short) length;
		else dev_2d_imagepointer[ i ] = 0;
		//else dev_2d_imagepointer[ i ] = 0.0f;

		//dev_2d_imagepointer[ i ] = length;
		
}


__global__ void fill_dos( float * imagepointer)
{
	int i = blockDim.x * blockIdx.x +threadIdx.x;
	imagepointer[i]=2.0f;
}